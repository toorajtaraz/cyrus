#include "hip/hip_runtime.h"
#include <api.h>
#include <interpolating_lhe.h>
#include <iostream>
#include <lhe.h>

double*** d_dp_luts = NULL;
long unsigned int N_DP_LUTS_I = 0;
long unsigned int N_DP_LUTS_J = 0;

void
clean_up()
{
  if (d_dp_luts != NULL) {
    for (int i = 0; i < N_DP_LUTS_I; i++) {
      for (int j = 0; j < N_DP_LUTS_J; j++) {
        delete[] d_dp_luts[i][j];
      }
      delete[] d_dp_luts[i];
    }
    delete[] d_dp_luts;
  }
}

cv::Mat
interpolating_lhe_api(cv::Mat src,
                      int window,
                      long long* taken_time_pure,
                      long long* taken_time_total)
{
  auto start = std::chrono::high_resolution_clock::now();
  cv::cuda::GpuMat d_src;
  d_src.upload(src);
  cv::Mat h_result(src.size().height, src.size().width, src.type());
  cv::cuda::GpuMat d_result(src.size().height, src.size().width, src.type());
  int offset = (int)floor((double)window / 2.0);
  int width = src.rows;
  int height = src.cols;
  int max_i = height + (offset - (height % offset));
  int max_j = width + (offset - (width % offset));

  int x_max = max_i / offset;
  int y_max = max_j / offset;
  if (d_dp_luts == NULL) {
    N_DP_LUTS_I = (max_i / offset);
    N_DP_LUTS_J = (max_j / offset);
    CUDA_CHECK(hipMallocManaged((void**)&d_dp_luts,
                                 sizeof(double**) * (max_i / offset)));
    for (int i = 0; i <= (max_i / offset); i++) {
      CUDA_CHECK(hipMallocManaged((void**)&(d_dp_luts[i]),
                                   sizeof(double*) * (max_j / offset)));
      for (int j = 0; j <= (max_j / offset); j++) {
        CUDA_CHECK(
          hipMallocManaged((void**)&(d_dp_luts[i][j]), sizeof(double) * 256));
      }
    }
  }
  dim3 dimBlock(32, 32, 1);
  dim3 dimGrid((d_src.cols * 2) / 32 + 2, (d_src.rows * 2) / 32, 1);
  auto start_gpu = std::chrono::high_resolution_clock::now();

  lhe_build_luts<<<1, 4>>>(
    d_dp_luts, d_src.data, offset, width, height, d_src.channels(), d_src.step);

  hipDeviceSynchronize();

  apply_interpolating_lhe<<<dimGrid, dimBlock>>>(d_result.data,
                                                 d_src.data,
                                                 window,
                                                 offset,
                                                 width,
                                                 height,
                                                 d_src.channels(),
                                                 d_src.step,
                                                 d_dp_luts);

  hipDeviceSynchronize();
  auto end_gpu = std::chrono::high_resolution_clock::now();
  *taken_time_pure =
    std::chrono::duration_cast<std::chrono::milliseconds>(end_gpu - start_gpu)
      .count();
  d_result.download(h_result);
  // free gpumat
  d_src.release();
  d_result.release();
  auto end = std::chrono::high_resolution_clock::now();
  *taken_time_total =
    std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
  return h_result;
}

cv::Mat
lhe_api(cv::Mat src, int window, long long* taken_time)
{
  cv::cuda::GpuMat d_src;
  d_src.upload(src);
  cv::Mat h_result(src.size().height, src.size().width, src.type());
  cv::cuda::GpuMat d_result(src.size().height, src.size().width, src.type());
  auto start_gpu = std::chrono::high_resolution_clock::now();

  apply_lhe<<<1, 64>>>(d_result.data,
                       d_src.data,
                       window,
                       d_src.cols,
                       d_src.rows,
                       d_src.step,
                       d_src.channels());
  hipDeviceSynchronize();
  auto end_gpu = std::chrono::high_resolution_clock::now();
  *taken_time =
    std::chrono::duration_cast<std::chrono::milliseconds>(end_gpu - start_gpu)
      .count();
  d_result.download(h_result);

  // free gpumat
  d_src.release();
  d_result.release();
  return h_result;
}